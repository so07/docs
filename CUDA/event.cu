#include <iostream>
#include <hip/hip_runtime.h>

int main () {

   int num_elemnts = 1<<8;

   // allocate memory
   int * h_buffer = (int *) malloc(sizeof(int) * num_elemnts);

   int * d_buffer;
   hipMalloc((void **) &d_buffer, sizeof(int) * num_elemnts);

   hipEvent_t start, stop;

   // create events
   hipEventCreate(&start);
   hipEventCreate(&stop);

   // start recording
   hipEventRecord(start);

   hipMemcpy(h_buffer, d_buffer, sizeof(int) * num_elemnts, hipMemcpyHostToDevice);

   // end recording
   hipEventRecord(stop);

   // synchronize on event stop
   hipEventSynchronize(stop);

   float Etime; // Elapsed time in ms
   hipEventElapsedTime(&Etime, start, stop);

   // release event resources
   hipEventDestroy(start);
   hipEventDestroy(stop);

   std::cout << "Etime / ms = " << Etime << std::endl;

   return 0;
}
